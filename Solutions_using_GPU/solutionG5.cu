#include "hip/hip_runtime.h"
// G5
// Finding number of K-Cliques in an undirected graph
// Find made iterative, one thread per subtree

#include <iostream>
#include <algorithm>
#include <map>
#include <chrono>
#include <assert.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;
using namespace std::chrono;

// It will store graph like adjacency list.
int *v;
int *v_size;
int n, m, k;

// It will store the degree of each node 
__global__ void degree(int *e1, int *e2, int *d, int *v_size, int m)
{
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx < m)
    {
        int x = e1[idx], y = e2[idx];
        int *dx = &d[x], *dy = &d[y];
        atomicAdd(dx,1);
        atomicAdd(dy,1);
        atomicAdd(&v_size[x],1);
    }
}

__global__ void prefix_sum(int *v_size, int n)
{
    for(int i = 1; i < n; i++)
    {
        v_size[i] += v_size[i - 1];
    }
}

__global__ void adj(int *e1, int *e2, int *v, int *v_i, int *v_size, int m)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < m)
    {
        int x = e1[idx], y = e2[idx];
        // x is smaller than y
        int i = atomicAdd(&v_i[x], 1);
        i += v_size[x-1];
        assert(i<m);
        v[i]=y;
    }
    
}


__global__ void find_iterative(int *d_k, bool *G_linear, int *imp, int *d_imp_size, int *cnt)
{
    int rootIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int imp_size = (*d_imp_size);
 
    if (rootIdx>=imp_size)return;
 
    int k = (*d_k);
    int root = imp[rootIdx];
    int thread_count=0;
    int lvl = 2;
    bool** lvl_vertices = (bool**)malloc((k + 1) * sizeof(bool*));
    int* num_lvl_vertices = (int*)malloc((k + 1) * sizeof(int));
    int* cur_vertex_id = (int*)malloc((k + 1) * sizeof(int));

    // The part of G_linear from root*imp_size to root*imp_size + imp_size - 1;
    lvl_vertices[lvl] = &(G_linear[rootIdx * imp_size]);
    
    // printf("inside Kernel, root = %d \n", root);
    // for(int i = 0; i < imp_size; i++)
    //     printf("%d ", lvl_vertices[lvl][i]);
    // printf("\n");
   
    cur_vertex_id[lvl] = 0;
    while(cur_vertex_id[lvl] < imp_size)
    {
        // printf("root = %d, lvl = %d, cur_vertex_id = %d \n", root, lvl, cur_vertex_id[lvl]);
        
        if(lvl_vertices[lvl][cur_vertex_id[lvl]] == 0)
        {
            // printf("Continued from cur_vertex_id = %d \n", cur_vertex_id[lvl]);
            cur_vertex_id[lvl]++;
            while(cur_vertex_id[lvl] == imp_size && lvl > 2)
            {
                // Go to parent level
                lvl = lvl - 1;

                // Go to parent's next sibling
                cur_vertex_id[lvl]++;
            }
            continue;
        }
        
        // vertex = imp[cur_vertex_id[lvl]];
        // vertex's adjacency list is the part of G_linear from vertex_id * imp_size to vertex_id * imp_size + imp_size - 1
        bool* adj_vertex = (bool*)malloc(imp_size * sizeof(bool));
        adj_vertex = &(G_linear[cur_vertex_id[lvl] * imp_size]);
            
        // printf("Till now ->\t\t\t");
        // for(int i = 0; i < imp_size; i++)
        //     printf("%d ", lvl_vertices[lvl][i]);
        // printf("\n");
        // printf("Adjacency list ->\t");
        // for(int i = 0; i < imp_size; i++)
        //     printf("%d ", adj_vertex[i]);
        // printf("\n");
            
        // intersec of adj_vertex[] with lvl_vertices[lvl][]
        lvl_vertices[lvl + 1] = (bool*)malloc(imp_size * sizeof(bool));
        num_lvl_vertices[lvl + 1] = 0;
        for(int i = 0; i < imp_size; i++)
        {
            lvl_vertices[lvl + 1][i] = (lvl_vertices[lvl][i] & adj_vertex[i]);
            if(lvl_vertices[lvl + 1][i] == 1)
            {
                num_lvl_vertices[lvl + 1]++; 
            }
        }
        
        // printf("Intersection ->\t\t");
        // for(int i = 0; i < imp_size; i++)
        //     printf("%d ", lvl_vertices[lvl + 1][i]);
        // printf("\n");
        // printf("num_lvl_vertices = %d \n \n", num_lvl_vertices[lvl + 1]);
        
        if(num_lvl_vertices[lvl + 1] > 0 && lvl + 1 < k)
        {
            lvl++;
            cur_vertex_id[lvl] = 0;
        }
        else
        {
            if(lvl + 1 == k)
            {
                thread_count+=num_lvl_vertices[lvl + 1]; 
            }

            // Go to next sibling
            cur_vertex_id[lvl]++;

            while(cur_vertex_id[lvl] == imp_size && lvl > 2)
            {
                // Go to parent level
                lvl = lvl - 1;

                // Go to parent's next sibling
                cur_vertex_id[lvl]++;
            }
        }
    }
    
    cnt[rootIdx] = thread_count;
    //printf("\nTotal count = %d\n--------------------------------------\n\n", thread_count);
}

int main()
{
    #ifndef ONLINE_JUDGE
    freopen("./input.txt", "r", stdin);
    //freopen("output.txt", "w", stdout);
    #endif

//--------------------------- INPUT Starts -----------------------------> 
    // First line of input should contain number of edges m and size of clique k.
    scanf("%d %d", &m, &k);

    n = 0;
    // map to remove duplicate edges
    map<pair<int,int>,int> mp; 
    for(int i=0; i<m; i++)
    {
        int x,y;
        scanf("%d %d", &x, &y);
        // x must smaller than y
        if(x > y) swap(x,y);
        if(x != y) mp[{x,y}] = 1;
        n = max(n, y);
    }
    n++;
    m = mp.size();

    // Storing unique edges in e1[i] - e2[i] 
    int *e1 = (int*) malloc(m * sizeof(int));
    int *e2 = (int*) malloc(m * sizeof(int));
    int i = 0;
    for(auto x: mp)
    {
        e1[i] = x.first.first;
        e2[i] = x.first.second;
        i++;
    }

    // edges in device
    int *d_e1, *d_e2;
    hipMalloc(&d_e1, m*sizeof(int));
    hipMalloc(&d_e2, m*sizeof(int));
    cudaCheckErrors("hipMalloc edges failure");
    hipMemcpy(d_e1, e1, m*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_e2, e2, m*sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy edges failure");

//--------------------------- INPUT Ends -------------------------------> 

//------------------------ ALGORITHM Starts ----------------------------> 
    // Start Time
    auto start_time = high_resolution_clock::now();

    // degree of nodes in device
    int *d_d, *d_v_size;
    hipMalloc(&d_d, n*sizeof(int));
    hipMalloc(&d_v_size, n*sizeof(int));
    cudaCheckErrors("hipMalloc degree failure");
    hipMemset(d_d, 0, n*sizeof(int));
    hipMemset(d_v_size, 0, n*sizeof(int));
    cudaCheckErrors("hipMemset degree failure");

    int deg_block_sz = 256;
    degree<<<(m+deg_block_sz-1)/deg_block_sz, deg_block_sz>>>(d_e1, d_e2, d_d, d_v_size, m);
    cudaCheckErrors("Kernel degree launch failure");
    prefix_sum<<<1,1>>>(d_v_size, n);
    cudaCheckErrors("Kernel prefix_sum launch failure");

    int d[n];
    v_size = (int*) malloc(n * sizeof(int));
    hipMemcpy(d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(v_size, d_v_size, n*sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy degree failure");
    
    int *d_v, *d_v_i;
    hipMalloc(&d_v, m*sizeof(int));
    hipMalloc(&d_v_i, n*sizeof(int));
    cudaCheckErrors("hipMalloc adjacency_matrix failure");
    hipMemset(d_v_i, 0, n*sizeof(int));
    cudaCheckErrors("hipMemset adjacency_matrix failure");

    adj<<<(m+deg_block_sz-1)/deg_block_sz, deg_block_sz>>>(d_e1, d_e2, d_v, d_v_i, d_v_size, m);
    cudaCheckErrors("Kernel adjacency_matrix launch failure");

    v = (int*) malloc(m * sizeof(int));
    hipMemcpy(v, d_v, m*sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy adjacency_matrix failure");

    // cout<<"printing degrees\n";
    // for(int i=0;i<n;i++)
    // {
    //     cout<<i<<" "<<d[i]<<"\n";
    // }
    // cout<<"\n";

    // Only those nodes will form k-clique that have degree >= k-1.
    int imp_size = 0;
    for(int i = 0; i < n; i++)
    {
        if(d[i] >= k - 1)
            imp_size++;
    }
    int *imp = (int*) calloc(imp_size, sizeof(int));
    for(int i = 0, j = 0; i < n; i++)
    {
        if(d[i] >= k - 1)
        {
            imp[j] = i;
            j++;
        }
    }
    
    cout<<"Printing important vec of size = "<< imp_size<<endl; 
    // for(int i=0;i<imp_size;i++)
    // {
    //     cout<<imp[i]<<" ";
    // }
    // cout<<"\n\n";

    /*
    need imp, imp_size, k, i, cnt, v, v_size in gpu memory 
    d_v_size and d_v are already in gpu memory
    remaining imp, imp_size, k, i and cnt
    */

    int *cnt = (int*)malloc(imp_size*sizeof(int));
    int *d_imp, *d_imp_size, *d_k, *d_cnt;
    hipMalloc(&d_imp, imp_size*sizeof(int));
    hipMalloc(&d_imp_size, sizeof(int));
    hipMalloc(&d_k, sizeof(int));
    hipMalloc(&d_cnt, imp_size*sizeof(int));
    cudaCheckErrors("hipMalloc failure");
    hipMemset(d_cnt, 0, imp_size*sizeof(int));
    cudaCheckErrors("hipMemset failure");
    hipMemcpy(d_imp, imp, imp_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_imp_size, &imp_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_k, &k, sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy failure");
    
    // making G linear
    bool *G_linear = (bool*)malloc(imp_size * imp_size * sizeof(bool));
    for(int i = 0; i < imp_size; i++)
    {
        int vertex1 = imp[i];
        for(int j = 0; j < imp_size; j++)
        {
            int vertex2 = imp[j];
            G_linear[i*imp_size + j] = (mp.find({vertex1, vertex2}) != mp.end());
        }
    }
    
    // printf("Printing G_linear \n");
    // for(int i = 0; i < imp_size * imp_size; i++)
    // {
    //     if(i%imp_size==0)cout<<"\n";
    //     printf("%d ", G_linear[i]);
    // }
    // printf("\n \n");

    // storing G_linear in gpu
    bool *d_G_linear;
    hipMalloc(&d_G_linear, imp_size * imp_size * sizeof(bool));
    cudaCheckErrors("hipMalloc G_linear failure");
    hipMemcpy(d_G_linear, G_linear, imp_size * imp_size * sizeof(bool), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy G_linear failure");

    
    cout<<"start "<<(imp_size+1023)/1024<<endl;
    find_iterative<<<(imp_size+1023)/1024,1024>>>(d_k, d_G_linear, d_imp, d_imp_size, d_cnt);

    // End Time
    auto end_time = high_resolution_clock::now();
//------------------------ ALGORITHM Ends ----------------------------> 

//------------------------ OUTPUT Starts -----------------------------> 

    hipMemcpy(cnt, d_cnt, imp_size*sizeof(int), hipMemcpyDeviceToHost);
    long long ans=0;
    for(int i=0;i<12;i++)
    {
        ans+=cnt[i];
        cout<<cnt[i]<<"\n";
    }
    

    // Calculating time duration.
    auto duration = duration_cast<microseconds> (end_time - start_time);
    float time_us = duration.count();
    float time_ms = (float) duration.count() / 1000;
    float time_s = (float) duration.count() / 1000000;
    
    printf("%ld \n", ans);
    printf("Time Taken -> \n");
    printf("%.3f seconds \n", time_s);
    printf("%.3f milliseconds \n", time_ms);
    printf("%.3f microseconds \n", time_us);
//------------------------- OUTPUT Ends ------------------------------> 

}
