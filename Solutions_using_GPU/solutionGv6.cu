// Finding number of K-Cliques in an undirected graph
// Only parallelizing v6 for degree and adjacency matrix


#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <map>
#include <chrono>
#include <assert.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;
using namespace std::chrono;

// It will store graph like adjacency list.
int *v;
int *v_size;
int n, m, k, cnt;

// It will recurse and find all possible K-Cliques and increment cnt if a K-Clique is found.
void find(int i, int options[], int options_size)
{
    //for(int i1 = 0; i1 < options_size; i1++)
    //    cout<<options[i1]<<" ";
    //cout<<endl;
    if(k-i+1 > options_size) return;
    if(i == k)
    {
        cnt += options_size;
        return;
    }

    for(int i1 = 0; i1 < options_size; i1++)
    {
        int x = options[i1];
        
        // Finding intersection of options and v[x]
        int intersec_size = 0, vsz = v_size[x] - v_size[x-1];
        for(int i2 = 0; i2 < vsz; i2++)
        {
            int nd = v[v_size[x-1]+i2];
            for(int i3=0;i3<options_size;i3++)
            {
                if(options[i3]==nd)
                    intersec_size++;
            }
            //if(binary_search(options, options + options_size, nd))
            //    intersec_size++;
        }

        int *intersec = (int*) malloc(intersec_size * sizeof(int));
        for(int i2 = 0, j = 0; i2 < vsz; i2++)
        {
            int nd = v[v_size[x-1]+i2];
            for(int i3=0;i3<options_size;i3++)
            {
                if(options[i3]==nd)
                {
                    intersec[j] = nd;
                    j++;
                }
            }
            //if(binary_search(options, options + options_size, nd))
            //{
            //    intersec[j] = nd;
            //    j++;
            //}
        }

        // Recursion
        //cout<<x<<endl;
        find(i+1, intersec, intersec_size);
    }
}

// It will store the degree of each node 
__global__ void degree(int *e1, int *e2, int *d, int *v_size, int m)
{
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx < m)
    {
        //printf("index = %d\n",idx);
        int x = e1[idx], y = e2[idx];
        // x is smaller than y
        //printf("index = %d\tx = %d\ty = %d\n", idx,x,y);
     
        int *dx = &d[x], *dy = &d[y];
        atomicAdd(dx,1);
        atomicAdd(dy,1);
        atomicAdd(&v_size[x],1);
    }
    
}

__global__ void prefix_sum(int *v_size, int n)
{
    for(int i=1;i<n;i++)
    {
        v_size[i]+=v_size[i-1];
    }
}

__global__ void adj(int *e1, int *e2, int *v, int *v_i, int *v_size, int m)
{
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx < m)
    {
        //printf("index = %d\n",idx);
        int x = e1[idx], y = e2[idx];
        // x is smaller than y
        int i=atomicAdd(&v_i[x],1);
        i+=v_size[x-1];
        //printf("index = %d\n", i);
        assert(i<m);
        v[i]=y;
    }
    
}

int main()
{
    #ifndef ONLINE_JUDGE
    freopen("./Test_Files/input103.txt", "r", stdin);
    freopen("output103Gv6.txt", "w", stdout);
    #endif

//--------------------------- INPUT Starts -----------------------------> 
    // First line of input should contain number of edges m and size of clique k.
    scanf("%d %d", &m, &k);

    n = 0;
    // map to remove duplicate edges
    map<pair<int,int>,int> mp; 
    for(int i=0; i<m; i++)
    {
        int x,y;
        scanf("%d %d", &x, &y);
        // x must smaller than y
        if(x > y) swap(x,y);
        if(x != y) mp[{x,y}] = 1;
        n = max(n, y);
    }
    n++;
    m = mp.size();
    // v = (int**) malloc(n * sizeof(int*));
    // v_size = (int*) calloc(n, sizeof(int));

    // Storing unique edges in e1[i] - e2[i] 
    int *e1 = (int*) malloc(m * sizeof(int));
    int *e2 = (int*) malloc(m * sizeof(int));
    int i = 0;
    for(auto x: mp)
    {
        e1[i] = x.first.first;
        e2[i] = x.first.second;
        i++;
    }

    //for(i=0;i<m;i++)
    //{
    //    cout<<e1[i]<<" "<<e2[i]<<endl;
    //}

    // edges in device
    int *d_e1, *d_e2;
    hipMalloc(&d_e1, m*sizeof(int));
    hipMalloc(&d_e2, m*sizeof(int));
    cudaCheckErrors("hipMalloc edges failure");
    hipMemcpy(d_e1, e1, m*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_e2, e2, m*sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy edges failure");

//--------------------------- INPUT Ends -------------------------------> 

//------------------------ ALGORITHM Starts ----------------------------> 
    // Start Time
    auto start_time = high_resolution_clock::now();

    // degree of nodes in device
    int *d_d, *d_v_size;
    hipMalloc(&d_d, n*sizeof(int));
    hipMalloc(&d_v_size, n*sizeof(int));
    cudaCheckErrors("hipMalloc degree failure");
    hipMemset(d_d, 0, n*sizeof(int));
    hipMemset(d_v_size, 0, n*sizeof(int));
    cudaCheckErrors("hipMemset degree failure");

    int deg_block_sz = 256;
    //cout<< "kernel config "<<(m+deg_block_sz-1)/deg_block_sz << deg_block_sz<<endl;
    degree<<<(m+deg_block_sz-1)/deg_block_sz, deg_block_sz>>>(d_e1, d_e2, d_d, d_v_size, m);
    cudaCheckErrors("Kernel degree launch failure");
    prefix_sum<<<1,1>>>(d_v_size, n);
    cudaCheckErrors("Kernel prefix_sum launch failure");
    // cudaDeviceSynchronize();

    int d[n];
    v_size = (int*) malloc(n * sizeof(int));
    hipMemcpy(d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(v_size, d_v_size, n*sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy degree failure");
    
    //for(int i = 0; i < n; i++)
    //    cout<<i<<" "<< d[i]<<endl;

    // Finding adjacency list v[] of graph
    // for(int i = 0; i < n; i++)
    //     v[i] = (int*)malloc(v_size[i] * sizeof(int));

    // int *v_i = (int*)calloc(n, sizeof(int));
    // for(auto it: mp)
    // {
    //     pair<int,int> p = it.first;
    //     int x = p.first, y = p.second;
    //     // x is smaller than y
    //     v[x][v_i[x]] = y;
    //     v_i[x]++;
    // }

    int *d_v, *d_v_i;
    hipMalloc(&d_v, m*sizeof(int));
    hipMalloc(&d_v_i, n*sizeof(int));
    cudaCheckErrors("hipMalloc adjacency_matrix failure");
    hipMemset(d_v_i, 0, n*sizeof(int));
    cudaCheckErrors("hipMemset adjacency_matrix failure");

    adj<<<(m+deg_block_sz-1)/deg_block_sz, deg_block_sz>>>(d_e1, d_e2, d_v, d_v_i, d_v_size, m);
    cudaCheckErrors("Kernel adjacency_matrix launch failure");

    v = (int*) malloc(m * sizeof(int));
    hipMemcpy(v, d_v, m*sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy adjacency_matrix failure");


    // Only those nodes will form k-clique that have degree >= k-1.
    int imp_size = 0;
    for(int i = 0; i < n; i++)
    {
        if(d[i] >= k - 1)
            imp_size++;
    }

    int *imp = (int*) calloc(imp_size, sizeof(int));
    for(int i = 0, j = 0; i < n; i++)
    {
        if(d[i] >= k - 1)
        {
            imp[j] = i;
            j++;
        }
    }
 
    //for(int i = 0; i < imp_size; i++)
    //    cout<<i<<" "<< imp[i]<<endl;
    
    cnt=0;
    find(1, imp, imp_size);

    // End Time
    auto end_time = high_resolution_clock::now();
//------------------------ ALGORITHM Ends ----------------------------> 

//------------------------ OUTPUT Starts -----------------------------> 
    // Calculating time duration.
    auto duration = duration_cast<microseconds> (end_time - start_time);
    float time_us = duration.count();
    float time_ms = (float) duration.count() / 1000;
    float time_s = (float) duration.count() / 1000000;

    printf("%d \n", cnt);
    printf("Time Taken -> \n");
    printf("%.3f seconds \n", time_s);
    printf("%.3f milliseconds \n", time_ms);
    printf("%.3f microseconds \n", time_us);
//------------------------- OUTPUT Ends ------------------------------> 

}
