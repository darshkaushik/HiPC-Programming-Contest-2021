// Finding number of K-Cliques in an undirected graph
// Only parallelizing v6 for degree and v_size


#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <map>
#include <chrono>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;
using namespace std::chrono;

// It will store graph like adjacency list.
int **v;
int *v_size;
int n, m, k, cnt;

// It will recurse and find all possible K-Cliques and increment cnt if a K-Clique is found.
void find(int i, int options[], int options_size)
{
    if(k-i+1 > options_size) return;
    if(i == k)
    {
        cnt += options_size;
        return;
    }

    for(int i1 = 0; i1 < options_size; i1++)
    {
        int x = options[i1];
        
        // Finding intersection of options and v[x]
        int intersec_size = 0;
        for(int i2 = 0; i2 < v_size[x]; i2++)
        {
            int nd = v[x][i2];
            if(binary_search(options, options + options_size, nd))
                intersec_size++;
        }

        int *intersec = (int*) malloc(intersec_size * sizeof(int));
        for(int i2 = 0, j = 0; i2 < v_size[x]; i2++)
        {
            int nd = v[x][i2];
            if(binary_search(options, options + options_size, nd))
            {
                intersec[j] = nd;
                j++;
            }
        }

        // Recursion
        find(i+1, intersec, intersec_size);
    }
}

// It will store the degree of each node 
__global__ void degree(int *e1, int *e2, int *d, int *v_size, int m)
{
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx < m)
    {
        //printf("index = %d\n",idx);
        int x = e1[idx], y = e2[idx];
        // x is smaller than y
        //printf("index = %d\tx = %d\ty = %d\n", idx,x,y);
     
        int *dx = &d[x], *dy = &d[y];
        atomicAdd(dx,1);
        atomicAdd(dy,1);
        atomicAdd(&v_size[x],1);
    }
    
}

int main()
{
    #ifndef ONLINE_JUDGE
    freopen("./Test_Files/input001.txt", "r", stdin);
    freopen("output001Gv6.txt", "w", stdout);
    #endif

//--------------------------- INPUT Starts -----------------------------> 
    // First line of input should contain number of edges m and size of clique k.
    scanf("%d %d", &m, &k);

    n = 0;
    // map to remove duplicate edges
    map<pair<int,int>,int> mp; 
    for(int i=0; i<m; i++)
    {
        int x,y;
        scanf("%d %d", &x, &y);
        // x must smaller than y
        if(x > y) swap(x,y);
        if(x != y) mp[{x,y}] = 1;
        n = max(n, y);
    }
    n++;
    m = mp.size();
    v = (int**) malloc(n * sizeof(int*));
    v_size = (int*) calloc(n, sizeof(int));

    // Storing unique edges in e1[i] - e2[i] 
    int *e1 = (int*) malloc(m * sizeof(int));
    int *e2 = (int*) malloc(m * sizeof(int));
    int i = 0;
    for(auto x: mp)
    {
        e1[i] = x.first.first;
        e2[i] = x.first.second;
        i++;
    }

    //for(i=0;i<m;i++)
    //{
    //    cout<<e1[i]<<" "<<e2[i]<<endl;
    //}

    // edges in device
    int *d_e1, *d_e2;
    hipMalloc(&d_e1, m*sizeof(int));
    hipMalloc(&d_e2, m*sizeof(int));
    cudaCheckErrors("hipMalloc edges failure");
    hipMemcpy(d_e1, e1, m*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_e2, e2, m*sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy edges failure");

//--------------------------- INPUT Ends -------------------------------> 

//------------------------ ALGORITHM Starts ----------------------------> 
    // Start Time
    auto start_time = high_resolution_clock::now();

    // Print this to know the number of nodes and unique edges.
    // printf("%d %d", n, m);

    // d[i] will tell degree of node i.
    // int *d = (int*) calloc(n, sizeof(int));
    // for(auto it: mp)
    // {
    //     pair<int,int> p = it.first;
    //     int x = p.first, y = p.second;
    //     d[x]++;
    //     d[y]++;
    //     // x is smaller than y
    //     v_size[x]++;
    // }

    // degree of nodes in device
    int *d_d, *d_v_size;
    hipMalloc(&d_d, n*sizeof(int));
    hipMalloc(&d_v_size, n*sizeof(int));
    cudaCheckErrors("hipMalloc degree failure");
    hipMemset(d_d, 0, n*sizeof(int));
    hipMemset(d_v_size, 0, n*sizeof(int));
    cudaCheckErrors("hipMemset degree failure");

    int deg_block_sz = 256;
    //cout<< "kernel config "<<(m+deg_block_sz-1)/deg_block_sz << deg_block_sz<<endl;
    degree<<<(m+deg_block_sz-1)/deg_block_sz, deg_block_sz>>>(d_e1, d_e2, d_d, d_v_size, m);
    cudaCheckErrors("Kernel degree launch failure");
    // cudaDeviceSynchronize();
    
    int d[n];
    hipMemcpy(d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(v_size, d_v_size, n*sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy degree failure");
    
    //for(int i = 0; i < n; i++)
    //    cout<<i<<" "<< d[i]<<endl;

    // Finding adjacency list v[] of graph
    for(int i = 0; i < n; i++)
        v[i] = (int*)malloc(v_size[i] * sizeof(int));

    int *v_i = (int*)calloc(n, sizeof(int));
    for(auto it: mp)
    {
        pair<int,int> p = it.first;
        int x = p.first, y = p.second;
        // x is smaller than y
        v[x][v_i[x]] = y;
        v_i[x]++;
    }

    // Only those nodes will form k-clique that have degree >= k-1.
    int imp_size = 0;
    for(int i = 0; i < n; i++)
    {
        if(d[i] >= k - 1)
            imp_size++;
    }

    int *imp = (int*) calloc(imp_size, sizeof(int));
    for(int i = 0, j = 0; i < n; i++)
    {
        if(d[i] >= k - 1)
        {
            imp[j] = i;
            j++;
        }
    }
    
    cnt=0;
    find(1, imp, imp_size);

    // End Time
    auto end_time = high_resolution_clock::now();
//------------------------ ALGORITHM Ends ----------------------------> 

//------------------------ OUTPUT Starts -----------------------------> 
    // Calculating time duration.
    auto duration = duration_cast<microseconds> (end_time - start_time);
    float time_us = duration.count();
    float time_ms = (float) duration.count() / 1000;
    float time_s = (float) duration.count() / 1000000;

    printf("%d \n", cnt);
    printf("Time Taken -> \n");
    printf("%.3f seconds \n", time_s);
    printf("%.3f milliseconds \n", time_ms);
    printf("%.3f microseconds \n", time_us);
//------------------------- OUTPUT Ends ------------------------------> 

}
